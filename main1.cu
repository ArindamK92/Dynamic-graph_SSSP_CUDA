#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include<vector>
#include <set>
#include<queue>
#include <chrono> 
#include <thread>
#include<queue>
#include<stack>
#include<list>
#include "all_structures.h"
#include <thrust/find.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#define THREADS_PER_BLOCK 10 //we can change it

using namespace std;
using namespace std::chrono;

__global__ void initialize(int nodes, int src, RT_Vertex* SSSP, int* stencil)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < nodes)
	{
		if (index == src) { SSSP[index].Root = -1; }
		else { SSSP[index].Root = index; }
		SSSP[index].Level = 0;
		SSSP[index].Dist = 0.0;
		stencil[index] = index;
	}
}

__global__ void create_tree(Colwt2* cuda_adjlist_full_X, int start, int stop, RT_Vertex* SSSP, int src, int* d_affectedPointer, int numberofCudaThread)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int number_CudaThread = numberofCudaThread;
	
	if (index < number_CudaThread)
	{
		printf("source: %d", src);
		int y = cuda_adjlist_full_X[index + start].col;
		printf("y: %d", y);
		double mywt = cuda_adjlist_full_X[index + start].wt;

		SSSP[y].Parent = src; //mark the parent
		SSSP[y].EDGwt = mywt; //mark the edgewt
		SSSP[y].Level = 1; //mark the Level
		SSSP[y].Root = -1;
		SSSP[y].Dist = mywt;
		d_affectedPointer[y] = 1;
		printf("end if***");
		
	}

}

struct is_affected
{
	__host__ __device__
		bool operator()(const int x)
	{
		return (x == 1);
	}
};

int main() {

	double startx, endx, starty, endy;
	/***** Preprocessing to Graph (GUI) ***********/
	int nodes;
	printf("Enter number of total nodes: ");
	scanf("%d", &nodes);

	//List of Changes
	//There will be a list for inserts and a list for delete
	vector<xEdge> allChange;
	allChange.clear();

	/*** Read Remainder Edges as Graph ***/
	A_Network R;
	for (int i = 0; i < nodes; i++)
	{
		ADJ_Bundle adj_bundle;
		adj_bundle.Row = i;
		R.push_back(adj_bundle);
	}
	string file1 = "C:\\Users\\khand\\Desktop\\PhD\\CUDA test\\Test\\test 1\\fullGraph.txt";
	/*printf("Enter file1(Argv[1]) name: ");
	scanf("%s", &file1);*/
	char* cstr1 = &file1[0];
	readin_graphU(&R, nodes, cstr1);
	

	/*cout << "***R***" << endl;
	for (int i = 0; i < R.size(); i++)
	{
		cout << "row: " << R.at(i).Row << endl;
		for (int j = 0; j < R.at(i).ListW.size(); j++)
		{
			cout << "column: " << R.at(i).ListW.at(j).first << endl;
			cout << "weight: " << R.at(i).ListW.at(j).second << endl;
		}
	}*/

	int* key_R = new int[nodes]; //it stores the node. key is used to find the adj list of a specific node
	int* colStartPtr_R;
	hipMallocManaged(&colStartPtr_R, (nodes + 1) * sizeof(int)); //we take nodes +1 to store the start ptr of the first row 
	int total_adjmatrix_size_R = 0;
	colStartPtr_R[0] = 0;
	key_R[0] = R.at(0).Row;
	for (int i = 0; i < R.size(); i++)
	{
		key_R[i] = R.at(i).Row;
		int size = R.at(i).ListW.size();
		colStartPtr_R[i + 1] = colStartPtr_R[i] + size; //size of adjacency list per row is stored
		total_adjmatrix_size_R = total_adjmatrix_size_R + size;
	}

	Colwt2* cuda_adjlist_full_R;
	hipMallocManaged(&cuda_adjlist_full_R, total_adjmatrix_size_R * sizeof(Colwt2));
	for (int i = 0; i < R.size(); i++)
	{
		int colsize = R.at(i).ListW.size();

		for (int j = 0; j < colsize; j++)
		{
			cuda_adjlist_full_R[colStartPtr_R[i] + j].col = R.at(i).ListW.at(j).first;
			cuda_adjlist_full_R[colStartPtr_R[i] + j].wt = R.at(i).ListW.at(j).second;
		}
	}

	/*** Finished Reading CRT Tree **/

	/*** Read the Certificate ***/
	A_Network X;
	for (int i = 0; i < nodes; i++)
	{
		ADJ_Bundle adj_bundle;
		adj_bundle.Row = i;
		X.push_back(adj_bundle);
	}
	string file2 = "C:\\Users\\khand\\Desktop\\PhD\\CUDA test\\Test\\test 1\\SSSP.txt";
	/*printf("Enter file1(Argv[2]) name: ");
	scanf("%s", &file2);*/
	char* cstr2 = &file2[0];
	readin_network(&X, cstr2, -1);
	/*for (int i = 0; i < X.size(); i++)
	{
		cout << "row: " << X.at(i).Row << endl;
		for (int j = 0; j < X.at(i).ListW.size(); j++)
		{
			cout <<"column: "<< X.at(i).ListW.at(j).first << endl;
			cout <<"weight: "<< X.at(i).ListW.at(j).second << endl;
		}
		
	}*/


	int* key_X = new int[nodes]; //it stores the node. key is used to find the adj list of a specific node
	int* colStartPtr_X = new int[nodes + 1]; //we take nodes +1 to store the start ptr of the first row 
	int total_adjmatrix_size_X = 0;
	colStartPtr_X[0] = 0;
	key_X[0] = X.at(0).Row;
	for (int i = 0; i < X.size(); i++)
	{
		key_X[i] = X.at(i).Row;
		int size = X.at(i).ListW.size();
		colStartPtr_X[i + 1] = colStartPtr_X[i] + size; //size of adjacency list per row is stored
		total_adjmatrix_size_X = total_adjmatrix_size_X + size;
	}
	/*for (int i = 0; i < X.size()+1; i++)
	{
		cout << colStartPtr_X[i] << endl;
	}*/
	Colwt2* cuda_adjlist_full_X;
	hipMallocManaged(&cuda_adjlist_full_X, total_adjmatrix_size_X * sizeof(Colwt2));
	for (int i = 0; i < X.size(); i++)
	{
		int colsize = X.at(i).ListW.size();

		for (int j = 0; j < colsize; j++)
		{
			cuda_adjlist_full_X[colStartPtr_X[i] + j].col = X.at(i).ListW.at(j).first;
			cuda_adjlist_full_X[colStartPtr_X[i] + j].wt = X.at(i).ListW.at(j).second;
			/*cout <<"col: "<< cuda_adjlist_full_X[colStartPtr_X[i] + j].col << " weight: " << cuda_adjlist_full_X[colStartPtr_X[i] + j].wt << endl;*/
		}
	}

	/*** Finished Reading CRT Tree **/

	 /*** Read set of Changed Edges ***/
	string file3 = "C:\\Users\\khand\\Desktop\\PhD\\CUDA test\\Test\\test 1\\changeEdges.txt";
	/*printf("Enter file1(Argv[3]) name: ");
	scanf("%s", &file3);*/
	char* cstr3 = &file3[0];
	readin_changes(cstr3, &allChange);

	/*for (int i = 0; i < allChange.size(); i++)
	{
		cout <<"inst: "<< allChange.at(i).inst << endl;
		cout << "node1: " << allChange.at(i).theEdge.node1 << " node2: " << allChange.at(i).theEdge.node2 << " weight: " << allChange.at(i).theEdge.edge_wt << endl;
	}*/

	//new addition
	xEdge_cuda* allChange_cuda;
	int totalChange = allChange.size();
	hipMallocManaged(&allChange_cuda, totalChange * sizeof(xEdge_cuda));
	for (int i = 0; i < totalChange; i++)
	{
		allChange_cuda[i].node1 = allChange.at(i).theEdge.node1;
		allChange_cuda[i].node2 = allChange.at(i).theEdge.node2;
		allChange_cuda[i].edge_wt = allChange.at(i).theEdge.edge_wt;
		allChange_cuda[i].inst = allChange.at(i).inst;
		allChange_cuda[i].insertedToDatastructure = allChange.at(i).insertedToDatastructure;
	}
	/*** Finished Reading Changed Edges **/

	//Initializing  Rooted Tree
	RT_Vertex* SSSP;
	hipMallocManaged(&SSSP, nodes * sizeof(RT_Vertex));
	int* stencil; //stencil is used for tracking which node is being affected. 
	/*hipMallocManaged(&stencil, nodes * sizeof(int));*/
	hipMalloc((void**)&stencil, nodes * sizeof(int));
	int* stencil_c = new int[nodes];
	/*vector<SCC_vertex>SCC;*/
	int graphDirectedUndirectedIndicator = 0; // Should be 1 for SCC, 0 for not SCC. need to modify if we want SCC

	int source;
	printf("Enter source node: ");
	scanf("%d", &source);
	int p;

	if (graphDirectedUndirectedIndicator == 0) {
		int src = source; //the source from which the paths are computed
		initialize << <(nodes / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (nodes, src, SSSP, stencil); //kernet call
		hipDeviceSynchronize();
		hipMemcpy(stencil_c, stencil, nodes * sizeof(int), hipMemcpyDeviceToHost);
		/*for (int i = 0; i < nodes; i++)
		{
			cout << "stencil_c" << stencil_c[i] << endl;
		}*/
		/*for (int i = 0; i < nodes; i++)
		{

			cout <<"dist"<< SSSP->Dist << endl;
			cout <<"wt"<< SSSP->EDGwt << endl;
			cout << "level"<< SSSP->Root << endl;
			cout << "marked"<< SSSP->Parent << endl;
		}*/
		//Code for create_tree:
		int totalAffectedNode; //alias of numberOfAffectedNode

		int* affectedPointer;
		int* d_affectedPointer;
		hipMalloc((void**)&d_affectedPointer, nodes * sizeof(int));
		affectedPointer = (int*)calloc(nodes, sizeof(int));
		hipMemcpy(d_affectedPointer, affectedPointer, nodes * sizeof(int), hipMemcpyHostToDevice);
		/*hipMallocManaged(&affectedPointer, nodes * sizeof(int));*/

		totalAffectedNode = 1;
		int start = 0, end = 0;
		int* affected_nodes;
		affected_nodes = (int*)calloc(totalAffectedNode, sizeof(int));
		affected_nodes[0] = src;
		hipStream_t stream1;
		hipError_t result;
		result = hipStreamCreate(&stream1);
		while (totalAffectedNode > 0)
		{
			vector<int> affectedNodeAlias;
			for (int i = 0; i < totalAffectedNode; i++)
			{
				affectedNodeAlias.push_back(affected_nodes[i]);
			}
			for (int i = 0; i < totalAffectedNode; i++)
			{
				p = affectedNodeAlias.at(i);
				src = p;
				/*cout << "src: " << src << endl;*/
				start = colStartPtr_X[p];
				end = colStartPtr_X[p+1];
				int numberofCudaThread = end - start;
				/*cout << "adj node 4:" << cuda_adjlist_full_X[4].col<<endl;
				for (int i = 0; i < numberofCudaThread; i++)
				{
					cout << "adj node ptr" << i + start << endl;
					cout << "adj node"<<cuda_adjlist_full_X[i + start].col<<endl;
					cout << "adj node"<<cuda_adjlist_full_X[i + start].col<<endl;
				}*/
				create_tree << <(numberofCudaThread / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK, 0, stream1 >> > (cuda_adjlist_full_X, start, end, SSSP, src, d_affectedPointer, numberofCudaThread);
			}
			/*for (auto& t : exe_threads) t.join();*/
			hipStreamSynchronize(stream1);
			thrust::device_ptr<int> affectedPointer_alias(d_affectedPointer); // converting raw ptr to device_ptr
			hipMemcpy(affectedPointer, d_affectedPointer, nodes * sizeof(int), hipMemcpyDeviceToHost);
			/*for (int i = 0; i < nodes; i++)
			{
				cout << "after kernel call:" << affectedPointer[i] << endl;
			}*/
			/*thrust::device_ptr<int> affectedPointer_alias(affectedPointer);*/
			thrust::device_vector<int> affectedPointer_vector(affectedPointer_alias, affectedPointer_alias + nodes); //converting device_ptr to device_vector
			totalAffectedNode = thrust::count(affectedPointer_vector.begin(), affectedPointer_vector.end(), 1); //count the number of affected node
			cout << "totalAffectedNode: " << totalAffectedNode<<endl;
			affected_nodes = (int*)realloc(affected_nodes, totalAffectedNode * sizeof(int));
			/*affectedPointer = thrust::raw_pointer_cast(&affectedPointer_vector[0]);*/
			thrust::copy_if(thrust::host, stencil_c, stencil_c + nodes, affectedPointer, affected_nodes, is_affected());
			cout << "affected nodes " << endl;
			for (int i = 0; i < totalAffectedNode; i++)
			{
				cout << affected_nodes[i] << endl;
			}
			free(affectedPointer);
			affectedPointer = (int*)calloc(nodes, sizeof(int));
			hipMemcpy(d_affectedPointer, affectedPointer, nodes * sizeof(int), hipMemcpyHostToDevice);
			
		}

		free(affected_nodes);


		//****below code needs modification
		//create_treeS(&X, &R, &SSSP, src, p);


		//double maxV = (double)maxW * X.size();

		////Update the inserted and delted edges in the tree
		//int te = 0;
		//edge_update(&allChange, &X, &SSSP, &R, &maxV, &te, p);

		//rest_update(&X, &SSSP, &R, &maxV, &te, p);
		hipFree(d_affectedPointer);

	}
	else
	{
		//****below code needs modification
		/*SCC.clear();
		readin_SCC(argv[2], &SCC);
		update_SCC(&X, &SCC, &allChange);
		print_network(X);*/
	}

	hipFree(colStartPtr_R);
	hipFree(cuda_adjlist_full_R);
	hipFree(colStartPtr_X);
	hipFree(cuda_adjlist_full_X);
	hipFree(allChange_cuda);
	hipFree(SSSP);
	hipFree(stencil);

	return 0;
}

