#include "hip/hip_runtime.h"
/*extern "C" {} *///it will instruct the compiler to expect C linkage for your C functions, not C++ linkage.
//#include <thrust/find.h>
//#include <thrust/device_vector.h>
//#include <thrust/count.h>
//#include <thrust/copy.h>
//#include <thrust/execution_policy.h>
//#include <thrust/device_free.h>
#include <stdio.h>
#include "all_structure_cuda.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include<vector>
#include <chrono> 


#define THREADS_PER_BLOCK 1024 //we can change it

using namespace std;
using namespace std::chrono;

__global__ void initialize(int nodes, int src, RT_Vertex* SSSP)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < nodes)
	{
		if (index == src) { SSSP[index].Root = -1; } //try to avoid
		else { SSSP[index].Root = index; }
		SSSP[index].Dist = 0.0;
	}
}

__global__ void create_tree2(Colwt2* cuda_adjlist_full_X, int* d_colStartPtr_X, RT_Vertex* SSSP, int* d_affectedPointer, int* change_d, int nodes)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int flag = 0;
	
	if (index < nodes && d_affectedPointer[index] == 1)
	{
		
		int adjNodestartIndex = d_colStartPtr_X[index];
		for (int k = 0; k < d_colStartPtr_X[index + 1] - d_colStartPtr_X[index]; k++)
		{
			flag = 0;
			int y = cuda_adjlist_full_X[adjNodestartIndex + k].col;
			double mywt = cuda_adjlist_full_X[adjNodestartIndex + k].wt;
			if (mywt == -1) {
				flag = 1; }//invalid edge
			if (SSSP[y].Root == -1) { /*printf("check 1.2");*/ flag = 1; }
			if (SSSP[y].Root != y) { /*printf("check 1.3");*/ flag = 1; }
			if (flag == 0)
			{
				SSSP[y].Parent = index; //mark the parent
				SSSP[y].EDGwt = mywt; //mark the edgewt
				SSSP[y].Root = SSSP[index].Root;
				SSSP[y].Dist = SSSP[index].Dist + mywt;
				/*printf("mywt: %f", mywt);*/
				d_affectedPointer[y] = 1;
			}
		}
		d_affectedPointer[index] = 0;
		change_d[0] = 1;
	}
}

struct is_affected
{
	__host__ __device__
		bool operator()(const int x)
	{
		return (x == 1);
	}
};

__global__ void initializeUpdatedDist(double* d_UpdatedDist, RT_Vertex* SSSP, int X_size)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < X_size)
	{
		d_UpdatedDist[index] = SSSP[index].Dist;
	}
}

__global__ void initializeEdgedone(int* Edgedone, int totalChange)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < totalChange)
	{
		Edgedone[index] = -1;
	}
}

__global__ void insertDeleteEdge(xEdge_cuda* allChange_cuda, int* Edgedone, RT_Vertex* SSSP, int numS, int X_size, int* d_colStartPtr_X, Colwt2* cuda_adjlist_full_X, double* d_UpdatedDist, double inf, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < numS)
	{
		int node_1 = allChange_cuda[index].node1;
		int node_2 = allChange_cuda[index].node2;
		double edge_weight = allChange_cuda[index].edge_wt;

		if (node_1 > X_size || allChange_cuda[index].node2 > X_size)
		{
			Edgedone[index] = 0; //mark to not add
		}

		if (SSSP[node_1].Root != SSSP[node_2].Root)
		{
			Edgedone[index] = 0; //mark to not add
		}

		if (allChange_cuda[index].inst == 1)  //check x
		{
			//Check if edge exists--then dont insert 
			for (int k = 0; k < d_colStartPtr_X[node_1 + 1] - d_colStartPtr_X[node_1]; k++)
			{
				int myn = cuda_adjlist_full_X[d_colStartPtr_X[node_1] + k].col;
				double mywt = cuda_adjlist_full_X[d_colStartPtr_X[node_1] + k].wt; //check. added recently 01-15-20
				//****need check
				if (myn == node_2 && mywt <= edge_weight && mywt != -1)
				{
					Edgedone[index] = 0;
					break;
				}

			}//end of for
		}

		if (allChange_cuda[index].inst == 1 && Edgedone[index] != 0)
		{
			//We check the distances based on updateddist, to cull some insertion edges
			//In case of conflicts, actual distance remains correct

				//Default is remainder edge
			Edgedone[index] = 2;
			//Check twice once for  n1->n2 and once for n2->n1
			for (int yy = 0; yy < 2; yy++)
			{
				int node1, node2;
				if (yy == 0)
				{
					node1 = node_1;
					node2 = node_2;
				}
				else
				{
					node1 = node_2;
					node2 = node_1;
				}

		  //Check whether node1 is relaxed
				if (d_UpdatedDist[node2] > d_UpdatedDist[node1] + edge_weight)
				{
					//Update Parent and EdgeWt
					SSSP[node2].Parent = node1;
					SSSP[node2].EDGwt = edge_weight;
					d_UpdatedDist[node2] = d_UpdatedDist[node1] + edge_weight;
					SSSP[node2].Update = true;
					//Mark Edge to be added--node1 updated
					Edgedone[index] = 1;
					break;
				}

			}//end of for

		}//end of if insert

		//Deletion case
		//in case of deletion we don't update d_UpdatedDist
		if (allChange_cuda[index].inst == 0 && Edgedone[index] != 0)  //if deleted
		{
			Edgedone[index] = 3;
			//Check if edge exists in the tree
				//this will happen if node1 is parentof node or vice-versa
			bool iskeyedge = false;
			 //Mark edge as deleted
			if (SSSP[node_1].Parent == node_2)
			{
				//printf("YYY:%d:%d \n",mye.node1, mye.node2 );
				SSSP[node_1].EDGwt = inf;
				SSSP[node_1].Update = true;
				iskeyedge = true;
			}
			else {
				//Mark edge as deleted
				if (SSSP[node_2].Parent == node_1)
				{
					SSSP[node_2].EDGwt = inf;
					SSSP[node_2].Update = true;
					iskeyedge = true;
				}
			}


			//If  Key Edge Delete from key edges
		   //Set weights to -1;
			if (iskeyedge)
			{

				for (int k = 0; k < d_colStartPtr_X[node_1 + 1] - d_colStartPtr_X[node_1]; k++)
				{
					int myn = cuda_adjlist_full_X[d_colStartPtr_X[node_1] + k].col;
					if (myn == node_2)
					{
						cuda_adjlist_full_X[d_colStartPtr_X[node_1] + k].wt = -1; //set wt -1 in adj list of old sssp
						break;
					}

				}//end of for

				for (int k = 0; k < d_colStartPtr_X[node_2 + 1] - d_colStartPtr_X[node_2]; k++)
				{
					int myn = cuda_adjlist_full_X[d_colStartPtr_X[node_2] + k].col;
					if (myn == node_1)
					{
						cuda_adjlist_full_X[d_colStartPtr_X[node_2] + k].wt = -1; //set wt -1 in adj list of old sssp
						break;
					}

				}
			}

			for (int k = 0; k < colStartPtr_R[node_1 + 1] - colStartPtr_R[node_1]; k++)
			{
				int myn = cuda_adjlist_full_R[colStartPtr_R[node_1] + k].col;
				if (myn == node_2)
				{
					cuda_adjlist_full_R[colStartPtr_R[node_1] + k].wt = -1;
					break;
				}

			}

			for (int k = 0; k < colStartPtr_R[node_2 + 1] - colStartPtr_R[node_2]; k++)
			{
				int myn = cuda_adjlist_full_R[colStartPtr_R[node_2] + k].col;
				if (myn == node_1)
				{
					cuda_adjlist_full_R[colStartPtr_R[node_2] + k].wt = -1;
					break;
				}

			}
		}
	}
}

//The insertDeleteEdge method might connect wrong edge depending on the sequence when the edge was connected (mainly because of the synchronization related fault)
//We avoid this error by the below method without using locking approach
//The below method tries to fit the edges using relaxation step and if it can fit, a flag is raised. 
__global__ void checkInsertedEdges(int numS, int* Edgedone, double* d_UpdatedDist, xEdge_cuda* allChange_cuda, RT_Vertex* SSSP, int* change_d)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < numS)
	{

		if (Edgedone[index] == 1) //Edgedone will be 1 when edge is marked to be inserted
		{

			//get the edge
			int node_1 = allChange_cuda[index].node1;
			int node_2 = allChange_cuda[index].node2;
			double edgeWeight = allChange_cuda[index].edge_wt;
			//reset it to 0
			Edgedone[index] = 0;


			int node1, node2;
			if (d_UpdatedDist[node_1] > d_UpdatedDist[node_2])
			{
				node1 = node_1;
				node2 = node_2;
			}
			else
			{
				node1 = node_2;
				node2 = node_1;
			}

			//***Below two if logic will connect the correct edges.***
			//Check if some other edge was added--mark edge to be added //check x
			if (d_UpdatedDist[node1] > d_UpdatedDist[node2] + edgeWeight)
			{
				Edgedone[index] = 1;
			}

			//Check if correct edge wt was written--mark edge to be added //check x
			if ((SSSP[node1].Parent == node2) && (SSSP[node1].EDGwt > edgeWeight))
			{
				Edgedone[index] = 1;
			}


			if (Edgedone[index] == 1)
			{
				//Update Parent and EdgeWt
				SSSP[node1].Parent = node2;
				SSSP[node1].EDGwt = edgeWeight;
				d_UpdatedDist[node1] = d_UpdatedDist[SSSP[node1].Parent] + SSSP[node1].EDGwt;
				SSSP[node2].Update = true;
				change_d[0] = 1; //every time node dist is updated, the flag becomes 1
			}
		}
	}
}

//In the previous steps we do not update dist in SSSP structure, we update it in d_UpdatedDist structure in case of insertion.
//In the below method we update dist in SSSP structure
__global__ void updateDistance(int X_size, RT_Vertex* SSSP, double* d_UpdatedDist, double inf)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < X_size)
	{
		//do not update source node
		int px = SSSP[index].Parent;
		int flag = 0;
		if (SSSP[index].Parent == -1) { flag = 1; }


		if (flag != 1 && index == SSSP[px].Parent)
		{
			printf("DP: %d:%d %d:%d \n", index, SSSP[index].Parent, px, SSSP[px].Parent);
		}

		//For deletion case
		if (flag != 1 && SSSP[index].EDGwt == inf)
		{
			SSSP[index].Dist = inf;
			SSSP[index].Update = true;
			flag = 1;
		}

		//for insertion case
		if (flag != 1 && SSSP[index].Dist > d_UpdatedDist[index])
		{
			SSSP[index].Dist = d_UpdatedDist[index];
			SSSP[index].Update = true;
		}

	}
}
//In case of deletion we did not update distance in d_UpdatedDist till now.
//In the below method we update it
__global__ void initializeUpdatedDistOldDist(double* d_UpdatedDist, double* d_OldUpdate, RT_Vertex* SSSP, int X_size)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < X_size)
	{
		d_UpdatedDist[index] = SSSP[index].Dist; //**this will fill up the d_UpdatedDist values for deletion case also
		d_OldUpdate[index] = SSSP[index].Dist; //check if not need, we can omit it
	}
}


//1. This method tries to connect the disconnected nodes(disconnected by deletion) with other nodes using the original graph
//2. This method propagates the dist update till the leaf nodes
__global__ void updateNeighbors(double* d_UpdatedDist, RT_Vertex* SSSP, int X_size, /*int* d_mychange,*/ int* d_colStartPtr_X, Colwt2* cuda_adjlist_full_X, double inf, int* change_d, int its, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < X_size)
	{

		//If i is updated--update its neighbors as required
		if (SSSP[index].Update)
		{
			SSSP[index].Update = false;
			for (int j = 0; j < colStartPtr_R[index + 1] - colStartPtr_R[index]; j++)
			{
				int myn = cuda_adjlist_full_R[colStartPtr_R[index] + j].col;
				double mywt = cuda_adjlist_full_R[colStartPtr_R[index] + j].wt;

				if (SSSP[myn].EDGwt < mywt && SSSP[myn].Parent == index) //check if we have taken an edge with lower weight from the changeEdge set. if yes then don't update edgeweight
				{
					mywt = SSSP[myn].EDGwt;
				}
				//check if edge is deleted
				if (mywt < 0) { continue; } //if mywt = -1, that means node was deleted
				if (SSSP[index].Dist == inf)
				{
					if (myn == SSSP[index].Parent) //we don't update if myn is parent node
					{
						continue;
					}
					if (SSSP[myn].Parent == index) //if myn is not parent, then update dist to inf
					{
						d_UpdatedDist[myn] = inf;
						SSSP[myn].Dist = inf;
						SSSP[myn].Update = true;
						change_d[0] = 1;
						continue;
					}
					else {
						//below part connects disconnected(by deletion) nodes to other connected nodes
						if (SSSP[myn].Dist != inf) //if myn is from different branch, connect it with index when index.dist was inf
						{
							d_UpdatedDist[index] = d_UpdatedDist[myn] + mywt;
							SSSP[index].Dist = d_UpdatedDist[myn] + mywt;
							SSSP[index].Parent = myn;
							SSSP[index].EDGwt = mywt;
							SSSP[index].Update = true;
							change_d[0] = 1;
							continue;
						}
					}
				}
				if (SSSP[index].Dist != inf)
				{
					if (SSSP[myn].Dist == inf) //when myn is disconnected by a deletion, this code tries to connect it with other nodes
					{
						if (SSSP[index].Parent != myn)
						{
							d_UpdatedDist[myn] = d_UpdatedDist[index] + mywt;
							SSSP[myn].Dist = SSSP[index].Dist + mywt;
							SSSP[myn].EDGwt = mywt;
							SSSP[myn].Update = true;
							SSSP[myn].Parent = index;
							change_d[0] = 1;
							continue;
						}
						else {
							//don't do anything if myn is parent of index node
							continue;
						}

					}
					if (d_UpdatedDist[myn] > d_UpdatedDist[index] + mywt) //update both cases where parent of myn == index or parent of myn != index
					{
						d_UpdatedDist[myn] = d_UpdatedDist[index] + mywt;
						SSSP[myn].Dist = d_UpdatedDist[index] + mywt;
						SSSP[myn].Update = true;
						SSSP[myn].Parent = index;
						change_d[0] = 1;
						continue;
					}
					else
					{
						if (SSSP[myn].Parent == index)
						{
							d_UpdatedDist[myn] = d_UpdatedDist[index] + mywt;
							SSSP[myn].Dist = d_UpdatedDist[index] + mywt;
							SSSP[myn].Update = true;
							change_d[0] = 1;
							continue;
						}
						if ((d_UpdatedDist[index] > d_UpdatedDist[myn] + mywt) /*&& (SSSP[myn].Parent != index)*/)
						{
							d_UpdatedDist[index] = d_UpdatedDist[myn] + mywt;
							SSSP[index].Dist = d_UpdatedDist[myn] + mywt;
							SSSP[index].Update = true;
							SSSP[index].Parent = myn;
							change_d[0] = 1;
						}
					}
				}
			}
		}
	}
}

//doublecheck if update flag should be raised. Might be omitted.
__global__ void checkIfDistUpdated(int X_size, double* d_OldUpdate, double* d_UpdatedDist, RT_Vertex* SSSP)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < X_size)
	{
		if (d_OldUpdate[index] != d_UpdatedDist[index])
		{
			d_OldUpdate[index] = d_UpdatedDist[index];
			SSSP[index].Update = true;
		}
		else { SSSP[index].Update = false; }
	}
}

__global__ void updateDistanceFinal(int X_size, double* d_UpdatedDist, RT_Vertex* SSSP, double inf)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < X_size)
	{
		int flag = 0;
		//do not update parent
		if (SSSP[index].Parent == -1) { flag = 1; }

		if (flag == 0)
		{
			int px = SSSP[index].Parent;
			if (px > -1)
			{
				//printf("XX %d :%d \n", i, px);  
				if (index == SSSP[px].Parent)
				{
					printf("BBP %d %d \n", index, px);
				}
			}
			if (d_UpdatedDist[index] >= inf)
			{
				SSSP[index].Dist = inf;
			}
			else
			{
				SSSP[index].Dist = d_UpdatedDist[SSSP[index].Parent] + SSSP[index].EDGwt;
			}
		}
	}
}

void edge_update(int* totalChange, int* X_size, int* SSSP_size, xEdge_cuda* allChange_cuda, Colwt2* cuda_adjlist_full_X, int* colStartPtr_X, RT_Vertex* SSSP, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R, int* te, int* nodes);
void rest_update(int* X_size, Colwt2* cuda_adjlist_full_X, int* colStartPtr_X, RT_Vertex* SSSP, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R, int* nodes);


/*
1st arg: original graph file name
2nd arg: input SSSP file name
3rd arg: change edges file name
4th arg: no. of nodes
5th arg: no. of edges
****main commands to run****
nvcc -o op main2.cu
./op <fullgraph file name> <SSSP file name> <changeEdges file name> <no. of nodes> <no. of edges * 2 (or total number of lines in fullgraph file)>
*/
int main(int argc, char* argv[]) {


	/***** Preprocessing to Graph (GUI) ***********/
	int nodes, edges;
	hipError_t cudaStatus;
	nodes = atoi(argv[4]); 
	edges = atoi(argv[5]); 


	/*** Read Full Graph ***/
	int* colStartPtr_R;
	cout << "success 1" << endl;
	cudaStatus = hipMallocManaged((void**)&colStartPtr_R, (nodes + 1) * sizeof(int)); //we take nodes +1 to store the start ptr of the first row 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	int total_adjmatrix_size_R = edges * 2; //e.g.= (0 1 wt1), (1 0 wt1) both are same edge, but both will be there
	Colwt2* cuda_adjlist_full_R;
	cudaStatus = hipMallocManaged(&cuda_adjlist_full_R, total_adjmatrix_size_R * sizeof(Colwt2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	readin_graphU4(colStartPtr_R, cuda_adjlist_full_R, argv[1], &nodes);
	
	/*** Read the input SSSP ***/
	int* colStartPtr_X = (int*)malloc((nodes + 1) * sizeof(int));//we take nodes +1 to store the start ptr of the first row 
	int total_adjmatrix_size_X = (nodes - 1) * 2; //maximum number of edges in SSSP tree = nodes - 1. Each edge will take 2 places in adjacent list
	Colwt2* cuda_adjlist_full_X;
	cudaStatus = hipMallocManaged(&cuda_adjlist_full_X, total_adjmatrix_size_X * sizeof(Colwt2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at SSSP file");
	}

	readin_graphU4(colStartPtr_X, cuda_adjlist_full_X, argv[2], &nodes); //when cmd line arg used

	/*** Read the change file ***/
	vector<xEdge> allChange;
	allChange.clear();
	readin_changes(argv[3], &allChange);

	xEdge_cuda* allChange_cuda;
	int totalChange = allChange.size();
	cudaStatus = hipMallocManaged(&allChange_cuda, totalChange * sizeof(xEdge_cuda));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at changeEdge");
	}
	for (int i = 0; i < totalChange; i++)
	{
		allChange_cuda[i].node1 = allChange.at(i).theEdge.node1;
		allChange_cuda[i].node2 = allChange.at(i).theEdge.node2;
		allChange_cuda[i].edge_wt = allChange.at(i).theEdge.edge_wt;
		allChange_cuda[i].inst = allChange.at(i).inst;
	}

	//Initializing  Rooted Tree
	RT_Vertex* SSSP;
	cudaStatus = hipMallocManaged(&SSSP, nodes * sizeof(RT_Vertex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at SSSP structure");
	}
	
	int src = 0; //the source from which the paths are computed. Default we have taken 0 as source node
	initialize << <(nodes / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (nodes, src, SSSP); //kernet call
	hipDeviceSynchronize();

	//Code for create_tree from input SSSP
	auto startTime = high_resolution_clock::now(); //Time calculation
	int* affectedPointer;
	int* d_affectedPointer;
	cudaStatus = hipMalloc((void**)&d_affectedPointer, nodes * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at affectedPointer");
	}
	affectedPointer = (int*)calloc(nodes, sizeof(int));
	affectedPointer[0] = 1;
	hipMemcpy(d_affectedPointer, affectedPointer, nodes * sizeof(int), hipMemcpyHostToDevice);
	int* change_d = new int[1];
	int* change = new int[1];
	change[0] = 1;
	hipMalloc((void**)&change_d, 1 * sizeof(int));
	int* d_colStartPtr_X;
	hipMalloc((void**)&d_colStartPtr_X, (nodes + 1) * sizeof(int));
	hipMemcpy(d_colStartPtr_X, colStartPtr_X, (nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
	
	while (change[0] == 1)
	{
		change[0] = 0;
		hipMemcpy(change_d, change, 1 * sizeof(int), hipMemcpyHostToDevice);
		create_tree2 << <(nodes / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>> > (cuda_adjlist_full_X, d_colStartPtr_X, SSSP, d_affectedPointer, change_d, nodes);
		hipDeviceSynchronize();
		hipMemcpy(change, change_d, 1 * sizeof(int), hipMemcpyDeviceToHost);

	}

	free(affectedPointer);
	hipFree(d_affectedPointer);
	hipFree(d_colStartPtr_X);
	auto stopTime = high_resolution_clock::now();//Time calculation
	auto duration = duration_cast<microseconds>(stopTime - startTime);// Time calculation
	cout << "Time taken by create-tree function: "
		<< duration.count() << " microseconds" << endl;

	//test
	/*cout << "input sssp tree" << endl;
	for (int i = 0; i < nodes; i++)
	{
		cout << "node" << i << endl;
		cout << "dist" << SSSP[i].Dist << endl;
		cout << "parent" << SSSP[i].Parent << endl;
		cout << "Edgewt" << SSSP[i].EDGwt << endl;
	}*/
	//test end

	//edge_update function
	//Update the inserted and delted edges in the tree
	int x_size = nodes;
	int SSSP_size = nodes; //considering all nodes are participating in the SSSP
	int te = 0;
	auto startTime1 = high_resolution_clock::now();
	edge_update(&totalChange, &x_size, &SSSP_size, allChange_cuda, cuda_adjlist_full_X, colStartPtr_X, SSSP, cuda_adjlist_full_R, colStartPtr_R, &te, &nodes);
	cout << "after edge_update fn" << endl;
	//Time calculation
	auto stopTime1 = high_resolution_clock::now();
	// Time calculation
	auto duration1 = duration_cast<microseconds>(stopTime1 - startTime1);
	cout << "Time taken by edge_update function: "
		<< duration1.count() << " microseconds" << endl;

	auto startTime2 = high_resolution_clock::now();
	rest_update(&x_size, cuda_adjlist_full_X, colStartPtr_X, SSSP, cuda_adjlist_full_R, colStartPtr_R, &nodes);
	cout << "after rest_update fn" << endl;
	//Time calculation
	auto stopTime2 = high_resolution_clock::now();
	// Time calculation
	auto duration2 = duration_cast<microseconds>(stopTime2 - startTime2);
	cout << "Time taken by rest_update function: "
		<< duration2.count() << " microseconds" << endl;
	
	//Test code start
	/*cout << "SSSP" << endl;
	for (int i = 0; i < nodes; i++)
	{
		cout << "*******" << endl;
		cout << "node" << i << endl << "dist" << SSSP[i].Dist << endl << "parent" << SSSP[i].Parent << endl;
	}
	cout << "*******success*******" << endl;*/
	
	//Test code end


	hipFree(colStartPtr_R);
	hipFree(cuda_adjlist_full_R);
	hipFree(colStartPtr_X);
	hipFree(cuda_adjlist_full_X);
	hipFree(allChange_cuda);
	hipFree(SSSP);
	return 0;
}



void edge_update(int* totalChange, int* X_size, int* SSSP_size, xEdge_cuda* allChange_cuda, Colwt2* cuda_adjlist_full_X, int* colStartPtr_X, RT_Vertex* SSSP, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R, int* te, int* nodes)
{
	double inf = std::numeric_limits<double>::infinity();
	/*int* Edgedone;*/
	double* UpdatedDist;

	//Mark how the edge is processed
	int* Edgedone;
	hipMalloc((void**)&Edgedone, (*totalChange) * sizeof(int));
	//initialize Edgedone array with -1
	initializeEdgedone << <((*totalChange) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (Edgedone, *totalChange);
	hipDeviceSynchronize();

	//Store the updated distance value
	UpdatedDist = (double*)calloc(*X_size, sizeof(double));
	double* d_UpdatedDist;
	hipMalloc((void**)&d_UpdatedDist, (*X_size) * sizeof(double));
	hipMemcpy(d_UpdatedDist, UpdatedDist, (*X_size) * sizeof(double), hipMemcpyHostToDevice);

	//Initialize with current distance for each node
	initializeUpdatedDist << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (d_UpdatedDist, SSSP, *X_size);
	hipDeviceSynchronize();
	

	int numS = *totalChange;
	int* d_colStartPtr_X;
	hipMalloc((void**)&d_colStartPtr_X, (*nodes + 1) * sizeof(int));
	hipMemcpy(d_colStartPtr_X, colStartPtr_X, (*nodes + 1) * sizeof(int), hipMemcpyHostToDevice);

	insertDeleteEdge << < (numS / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (allChange_cuda, Edgedone, SSSP, numS, *X_size, d_colStartPtr_X, cuda_adjlist_full_X, d_UpdatedDist, inf, cuda_adjlist_full_R, colStartPtr_R);
	hipDeviceSynchronize();


	//Go over the inserted edges to see if they need to be changed. Correct edges are connected in this stage
	int* change_d = new int[1];
	int* change = new int[1];
	change[0] = 1;
	hipMalloc((void**)&change_d, 1 * sizeof(int));
	while (change[0] == 1)
	{
		change[0] = 0;
		hipMemcpy(change_d, change, 1 * sizeof(int), hipMemcpyHostToDevice);
		checkInsertedEdges << < (numS / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (numS, Edgedone, d_UpdatedDist, allChange_cuda, SSSP, change_d);
		hipDeviceSynchronize();
		hipMemcpy(change, change_d, 1 * sizeof(int), hipMemcpyDeviceToHost);

	}

	//Update the distances
	 //Initialize with current distance for each node
	updateDistance << < (numS / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (*X_size, SSSP, d_UpdatedDist, inf);
	hipDeviceSynchronize();

	hipFree(change_d);
	hipFree(d_UpdatedDist);
	hipFree(d_colStartPtr_X);
	free(UpdatedDist);
	return;
}


void rest_update(int* X_size, Colwt2* cuda_adjlist_full_X, int* colStartPtr_X, RT_Vertex* SSSP, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R, int* nodes)
{
	double inf = std::numeric_limits<double>::infinity();


	int its = 0; //number of iterations

	int* change_d = new int[1];
	int* change = new int[1]; //marking whether the connections changed in the iteration
	change[0] = 1;
	hipMalloc((void**)&change_d, 1 * sizeof(int));
	/*hipMemcpy(change_d, change, 1 * sizeof(int), hipMemcpyHostToDevice);*/ //recent change

	double* UpdatedDist;
	//Store the updated distance value
	UpdatedDist = (double*)calloc(*X_size, sizeof(double));
	double* d_UpdatedDist;
	hipMalloc((void**)&d_UpdatedDist, (*X_size) * sizeof(double));
	hipMemcpy(d_UpdatedDist, UpdatedDist, (*X_size) * sizeof(double), hipMemcpyHostToDevice);


	double* OldUpdate;
	//Store the old updated distance value
	OldUpdate = (double*)calloc(*X_size, sizeof(double));
	double* d_OldUpdate;
	hipMalloc((void**)&d_OldUpdate, (*X_size) * sizeof(double));
	hipMemcpy(d_OldUpdate, OldUpdate, (*X_size) * sizeof(double), hipMemcpyHostToDevice);

	int* d_colStartPtr_X;
	hipMalloc((void**)&d_colStartPtr_X, (*nodes + 1) * sizeof(int));
	hipMemcpy(d_colStartPtr_X, colStartPtr_X, (*nodes + 1) * sizeof(int), hipMemcpyHostToDevice);


	//Initialize with current distance for each node. mainly d_UpdatedDist is updated where deletion done
	initializeUpdatedDistOldDist << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (d_UpdatedDist, d_OldUpdate, SSSP, *X_size);
	hipDeviceSynchronize();

	while (change[0] == 1 && its < 70)
	{
		printf("Iteration:%d \n", its);

		change[0] = 0;
		hipMemcpy(change_d, change, 1 * sizeof(int), hipMemcpyHostToDevice);
		updateNeighbors << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (d_UpdatedDist, SSSP, *X_size, /*d_mychange,*/ d_colStartPtr_X, cuda_adjlist_full_X, inf, change_d, its, cuda_adjlist_full_R, colStartPtr_R);
		hipDeviceSynchronize();
		hipMemcpy(change, change_d, 1 * sizeof(int), hipMemcpyDeviceToHost);

	//Check if distance was updated
		//checkIfDistUpdated may not be required. Omit if not needed
		checkIfDistUpdated << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (*X_size, d_OldUpdate, d_UpdatedDist, SSSP);
		hipDeviceSynchronize();
		its++;
	}//end of while
	printf("Total Iterations to Converge %d \n", its);

	//Update the distances
	//Initialize with current distance for each node
	updateDistanceFinal << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (*X_size, d_UpdatedDist, SSSP, inf);
	hipDeviceSynchronize();

	free(UpdatedDist);
	free(OldUpdate);
	hipFree(change_d);
	hipFree(d_UpdatedDist);
	hipFree(d_OldUpdate);
	hipFree(d_colStartPtr_X);
	return;
}

