
#include <hip/hip_runtime.h>
///*extern "C" {} *///it will instruct the compiler to expect C linkage for your C functions, not C++ linkage.
//#include <stdio.h>
////#include "all_structures.h"
//#include "all_structure_cuda.cuh"
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//#include <iostream>
//#include <thrust/find.h>
//#include <thrust/device_vector.h>
//#include <thrust/count.h>
//#include <thrust/copy.h>
//#include <thrust/execution_policy.h>
//
//#include<vector>
////#include <set>
////#include<queue>
//#include <chrono> 
////#include <thread>
////#include<queue>
////#include<stack>
////#include<list>
//
//
//#define THREADS_PER_BLOCK 1024 //we can change it
//
//using namespace std;
//using namespace std::chrono;
//
//__global__ void initialize(int nodes, int src, RT_Vertex* SSSP, int* stencil)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//	if (index < nodes)
//	{
//		if (index == src) { SSSP[index].Root = -1; }
//		else { SSSP[index].Root = index; }
//		SSSP[index].Level = 0;
//		SSSP[index].Dist = 0.0;
//		stencil[index] = index;
//	}
//}
//
//__global__ void create_tree(Colwt2* cuda_adjlist_full_X, int start, RT_Vertex* SSSP, int src, int* d_affectedPointer, int numberofCudaThread)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//	int number_CudaThread = numberofCudaThread;
//	int flag = 0;
//
//	if (index < number_CudaThread)
//	{
//		/*printf("source: %d", src);*/
//		int y = cuda_adjlist_full_X[index + start].col;
//		/*printf("y: %d", y);*/
//		double mywt = cuda_adjlist_full_X[index + start].wt;
//		if (mywt == -1) { flag = 1; }//invalid edge
//		if (SSSP[y].Root == -1) { flag = 1; }
//		if (flag == 0)
//		{
//			SSSP[y].Parent = src; //mark the parent
//			SSSP[y].EDGwt = mywt; //mark the edgewt
//			SSSP[y].Level = SSSP[src].Level + 1; //mark the Level
//			SSSP[y].Root = SSSP[src].Root;
//			SSSP[y].Dist = SSSP[src].Dist + mywt;
//			d_affectedPointer[y] = 1;
//		}
//		/*printf("end if***");*/
//
//	}
//
//}
//
//
//struct is_affected
//{
//	__host__ __device__
//		bool operator()(const int x)
//	{
//		return (x == 1);
//	}
//};
//
//__global__ void initializeUpdatedDist(double* d_UpdatedDist, RT_Vertex* SSSP, int X_size)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//	if (index < X_size)
//	{
//		d_UpdatedDist[index] = SSSP[index].Dist;
//	}
//}
//
//__global__ void initializeEdgedone(int* Edgedone, int totalChange)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//	if (index < totalChange)
//	{
//		Edgedone[index] = -1;
//	}
//}
//
//__global__ void insertDeleteEdge(xEdge_cuda* allChange_cuda, int* Edgedone, RT_Vertex* SSSP, int numS, int X_size, int* d_colStartPtr_X, Colwt2* cuda_adjlist_full_X, double* d_UpdatedDist, double inf, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//	if (index < numS)
//	{
//		int node_1 = allChange_cuda[index].node1;
//		int node_2 = allChange_cuda[index].node2;
//		double edge_weight = allChange_cuda[index].edge_wt;
//
//		if (node_1 > X_size || allChange_cuda[index].node2 > X_size)
//		{
//			Edgedone[index] = 0; //mark to not add
//		}
//
//		if (SSSP[node_1].Root != SSSP[node_2].Root)
//		{
//			Edgedone[index] = 0; //mark to not add
//		}
//
//		if (allChange_cuda[index].inst == 1)  //check x
//		{
//			//Check if edge exists--then dont insert 
//			for (int k = 0; k < d_colStartPtr_X[node_1 + 1] - d_colStartPtr_X[node_1]; k++)
//			{
//				int myn = cuda_adjlist_full_X[d_colStartPtr_X[node_1] + k].col;
//				double mywt = cuda_adjlist_full_X[d_colStartPtr_X[node_1] + k].wt; //check. added recently 01-15-20
//				//****need check
//				if (myn == node_2 && mywt <= edge_weight && mywt != -1)
//				{
//					Edgedone[index] = 0;
//					break;
//				}
//
//			}//end of for
//		}
//
//		if (allChange_cuda[index].inst == 1 && Edgedone[index] != 0)
//		{
//			//We check the distances based on updateddist, to cull some insertion edges
//			//In case of conflicts, actual distance remains correct
//
//				//Default is remainder edge
//			Edgedone[index] = 2;
//			//Check twice once for  n1->n2 and once for n2->n1
//			for (int yy = 0; yy < 2; yy++)
//			{
//				int node1, node2;
//				if (yy == 0)
//				{
//					node1 = node_1;
//					node2 = node_2;
//				}
//				else
//				{
//					node1 = node_2;
//					node2 = node_1;
//				}
//
//				//  printf("%d:%f:::%d::%f:::%f \n", node1, UpdatedDist[node1],node2, UpdatedDist[node2], mye.edge_wt);
//		  //Check whether node1 is relaxed
//				if (d_UpdatedDist[node2] > d_UpdatedDist[node1] + edge_weight)
//				{
//					//Update Parent and EdgeWt
//					SSSP[node2].Parent = node1;
//					SSSP[node2].EDGwt = edge_weight;
//					d_UpdatedDist[node2] = d_UpdatedDist[node1] + edge_weight;
//					SSSP[node2].Update = true;
//					/*printf("@@@@node: %d, parent: %d, dist: %f", node2, SSSP[node2].Parent, d_UpdatedDist[node2]);*/
//					//Mark Edge to be added--node1 updated
//					Edgedone[index] = 1;
//					break;
//				}
//
//			}//end of for
//
//		}//end of if insert
//
//		//Deletion case
//		//in case of deletion we don't update d_UpdatedDist
//		if (allChange_cuda[index].inst == 0 && Edgedone[index] != 0)  //if deleted
//		{
//			Edgedone[index] = 3;
//			//Check if edge exists in the tree
//				//this will happen if node1 is parentof node or vice-versa
//			bool iskeyedge = false;
//
//			// printf("XXX:%d:%d \n",mye.node1, mye.node2 );
//
//					 //Mark edge as deleted
//			if (SSSP[node_1].Parent == node_2)
//			{
//				//printf("YYY:%d:%d \n",mye.node1, mye.node2 );
//				SSSP[node_1].EDGwt = inf;
//				SSSP[node_1].Update = true;
//				iskeyedge = true;
//				/*d_UpdatedDist[node_1] = inf;*/ //check. added recently 01-15-20
//			}
//			else {
//				//Mark edge as deleted
//				if (SSSP[node_2].Parent == node_1)
//				{
//					// printf("ZZZ:%d:%d \n",mye.node1, mye.node2 );
//					SSSP[node_2].EDGwt = inf;
//					SSSP[node_2].Update = true;
//					iskeyedge = true;
//					/*d_UpdatedDist[node_2] = inf;*/ //check. added recently 01-15-20
//				}
//			}
//
//
//			//If  Key Edge Delete from key edges
//		   //Set weights to -1;
//			if (iskeyedge)
//			{
//
//				for (int k = 0; k < d_colStartPtr_X[node_1 + 1] - d_colStartPtr_X[node_1]; k++)
//				{
//					////TEPS:
//					//*te = *te + 1;
//					int myn = cuda_adjlist_full_X[d_colStartPtr_X[node_1] + k].col;
//					if (myn == node_2)
//					{
//						cuda_adjlist_full_X[d_colStartPtr_X[node_1] + k].wt = -1; //set wt -1 in adj list of old sssp
//						break;
//					}
//
//				}//end of for
//
//				for (int k = 0; k < d_colStartPtr_X[node_2 + 1] - d_colStartPtr_X[node_2]; k++)
//				{
//					////TEPS:
//					//*te = *te + 1;
//					int myn = cuda_adjlist_full_X[d_colStartPtr_X[node_2] + k].col;
//					if (myn == node_1)
//					{
//						cuda_adjlist_full_X[d_colStartPtr_X[node_2] + k].wt = -1; //set wt -1 in adj list of old sssp
//						break;
//					}
//
//				}
//			}//end of if
//
//
//			/*else      // check. recently added 24-01-2020. The below part is required for all as we consider full graph
//			{*/
//
//			for (int k = 0; k < colStartPtr_R[node_1 + 1] - colStartPtr_R[node_1]; k++)
//			{
//				int myn = cuda_adjlist_full_R[colStartPtr_R[node_1] + k].col;
//				if (myn == node_2)
//				{
//					cuda_adjlist_full_R[colStartPtr_R[node_1] + k].wt = -1;
//					break;
//				}
//
//			}//end of for
//
//			for (int k = 0; k < colStartPtr_R[node_2 + 1] - colStartPtr_R[node_2]; k++)
//			{
//				int myn = cuda_adjlist_full_R[colStartPtr_R[node_2] + k].col;
//				if (myn == node_1)
//				{
//					cuda_adjlist_full_R[colStartPtr_R[node_2] + k].wt = -1;
//					break;
//				}
//
//			}//end of for
//
//		//}//end of if
//
//		}//end of else if deleted
//	}
//}
//
//
//__global__ void checkInsertedEdges(int numS, int* Edgedone, double* d_UpdatedDist, xEdge_cuda* allChange_cuda, RT_Vertex* SSSP, int* change_d)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//	if (index < numS)
//	{
//
//		if (Edgedone[index] == 1) //Edgedone will be 1 when edge is marked to be inserted
//		{
//
//			//get the edge
//			int node_1 = allChange_cuda[index].node1;
//			int node_2 = allChange_cuda[index].node2;
//			double edgeWeight = allChange_cuda[index].edge_wt;
//			//reset it to 0
//			Edgedone[index] = 0;
//
//
//			int node1, node2;
//			if (d_UpdatedDist[node_1] > d_UpdatedDist[node_2])
//			{
//				node1 = node_1;
//				node2 = node_2;
//			}
//			else
//			{
//				node1 = node_2;
//				node2 = node_1;
//			}
//
//			//Check if some other edge was added--mark edge to be added //check x
//			if (d_UpdatedDist[node1] > d_UpdatedDist[node2] + edgeWeight)
//			{
//				Edgedone[index] = 1;
//			}
//
//			//Check if correct edge wt was written--mark edge to be added //check x
//			if ((SSSP[node1].Parent == node2) && (SSSP[node1].EDGwt > edgeWeight))
//			{
//				Edgedone[index] = 1;
//			}
//
//
//			if (Edgedone[index] == 1)
//			{
//				//Update Parent and EdgeWt
//				SSSP[node1].Parent = node2;
//				SSSP[node1].EDGwt = edgeWeight;
//				d_UpdatedDist[node1] = d_UpdatedDist[SSSP[node1].Parent] + SSSP[node1].EDGwt;
//				SSSP[node2].Update = true;
//				change_d[0] = 1;
//			}
//
//
//		}//end of if
//	}
//}
//
//__global__ void updateDistance(int X_size, RT_Vertex* SSSP, double* d_UpdatedDist, double inf)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//	if (index < X_size)
//	{
//		//do not update source node
//		int px = SSSP[index].Parent;
//		int flag = 0;
//		if (SSSP[index].Parent == -1) { flag = 1; }
//
//
//		if (flag != 1 && index == SSSP[px].Parent)
//		{
//			printf("DP: %d:%d %d:%d \n", index, SSSP[index].Parent, px, SSSP[px].Parent);
//		}
//
//		//For deletion case
//		if (flag != 1 && SSSP[index].EDGwt == inf)
//		{
//			SSSP[index].Dist = inf;
//			SSSP[index].Update = true;
//			flag = 1;
//		}
//
//		//what is p and why the below code needed??
//		/*if (d_UpdatedDist[p] == *maxW)
//		{
//			SSSP->at(i).Dist = *maxW; SSSP->at(i).Update = true; continue;
//		}*/
//
//		//for insertion case
//		if (flag != 1 && SSSP[index].Dist > d_UpdatedDist[index])
//		{
//			SSSP[index].Dist = d_UpdatedDist[index];
//			/*printf("In updateDistance:  index: %d, dist:%f\n", index, SSSP[index].Dist);*/
//			SSSP[index].Update = true;
//		}
//
//	}
//}
//
//__global__ void initializeUpdatedDistOldDist(double* d_UpdatedDist, double* d_OldUpdate, RT_Vertex* SSSP, int X_size)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//	if (index < X_size)
//	{
//		d_UpdatedDist[index] = SSSP[index].Dist; //this will fill up the d_UpdatedDist values for deletion case also
//		d_OldUpdate[index] = SSSP[index].Dist;
//		/*printf("****Inside initializeUpdatedDistOldDist: %d edge weight: %f parent: %d dist: %f\n", index, SSSP[index].EDGwt, SSSP[index].Parent, SSSP[index].Dist);*/
//	}
//}
//
//
////revised function //check. recently added function. 24-01-2020
//__global__ void updateNeighbors(double* d_UpdatedDist, RT_Vertex* SSSP, int X_size, int* d_mychange, int* d_colStartPtr_X, Colwt2* cuda_adjlist_full_X, double inf, int* change_d, int its, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//	if (index < X_size)
//	{
//
//		//If i is updated--update its neighbors as required
//		if (SSSP[index].Update)
//		{
//			SSSP[index].Update = false;
//			for (int j = 0; j < colStartPtr_R[index + 1] - colStartPtr_R[index]; j++)
//			{
//				int myn = cuda_adjlist_full_R[colStartPtr_R[index] + j].col;
//				double mywt = cuda_adjlist_full_R[colStartPtr_R[index] + j].wt;
//
//				if (SSSP[myn].EDGwt < mywt && SSSP[myn].Parent == index) //check if we have taken an edge with lower weight from the changeEdge set. if yes then don't update edgeweight
//				{
//					mywt = SSSP[myn].EDGwt;
//				}
//
//				/*printf("****####index: %d col: %d  weight: %f\n", index, myn, mywt);*/
//				/*printf("****####index: %d edge weight: %f parent: %d dist: %f\n", index, SSSP[index].EDGwt, SSSP[index].Parent, SSSP[index].Dist);
//				printf("****myn: %d myn weight: %f parent: %d dist: %f\n", myn, mywt, SSSP[myn].Parent, SSSP[myn].Dist);*/
//
//				//check if edge is deleted
//				if (mywt < 0) { continue; } //if mywt = -1, that means node was deleted
//				if (SSSP[index].Dist == inf)
//				{
//					/*printf("$$$$: %d edge weight: %f parent: %d dist: %f\n", index, SSSP[index].EDGwt, SSSP[index].Parent, SSSP[index].Dist);
//					printf("$$$$myn: %d myn weight: %f parent: %d dist: %f\n", myn, mywt, SSSP[myn].Parent, SSSP[myn].Dist);
//					*/
//					if (myn == SSSP[index].Parent)
//					{
//						continue;
//					}
//					if (SSSP[myn].Parent == index)
//					{
//						d_UpdatedDist[myn] = inf;
//						SSSP[myn].Dist = inf;
//						SSSP[myn].Update = true;
//						change_d[0] = 1;
//						/*printf("&&&&index: %d edge weight: %f parent: %d dist: %f\n", index, SSSP[index].EDGwt, SSSP[index].Parent, SSSP[index].Dist);
//						printf("&&&&myn: %d myn weight: %f parent: %d dist: %f\n", myn, mywt, SSSP[myn].Parent, SSSP[myn].Dist);
//						*/
//						continue;
//					}
//					else {
//						if (SSSP[myn].Dist != inf)
//						{
//							d_UpdatedDist[index] = d_UpdatedDist[myn] + mywt;
//							SSSP[index].Dist = d_UpdatedDist[myn] + mywt;
//							SSSP[index].Parent = myn;
//							SSSP[index].EDGwt = mywt;
//							SSSP[index].Update = true;
//							change_d[0] = 1;
//							/*printf("++++index: %d edge weight: %f parent: %d dist: %f\n", index, SSSP[index].EDGwt, SSSP[index].Parent, SSSP[index].Dist);
//							printf("++++myn: %d myn weight: %f parent: %d dist: %f\n", myn, mywt, SSSP[myn].Parent, SSSP[myn].Dist);*/
//
//							continue;
//						}
//					}
//				}
//				if (SSSP[index].Dist != inf)
//				{
//					/*printf("Not inf: index: %d edge weight: %f parent: %d dist: %f\n", index, SSSP[index].EDGwt, SSSP[index].Parent, SSSP[index].Dist);
//					printf("Not infmyn: %d myn weight: %f parent: %d dist: %f\n", myn, mywt, SSSP[myn].Parent, SSSP[myn].Dist);
//					*/
//					if (SSSP[myn].Dist == inf)
//					{
//						if (SSSP[index].Parent != myn)
//						{
//							//if (SSSP[myn].EDGwt < mywt && (SSSP[myn].Parent == index)) //check if we have taken an edge with lower weight from the changeEdge set. if yes then don't update edgeweight
//							//{
//							//	mywt = SSSP[myn].EDGwt;
//							//}
//							d_UpdatedDist[myn] = d_UpdatedDist[index] + mywt;
//							SSSP[myn].Dist = SSSP[index].Dist + mywt;
//							SSSP[myn].EDGwt = mywt;
//							SSSP[myn].Update = true;
//							SSSP[myn].Parent = index;
//							change_d[0] = 1;
//							continue;
//						}
//						else {
//							//don't do anything if myn is parent of index node
//							continue;
//						}
//
//					}
//					if (d_UpdatedDist[myn] > d_UpdatedDist[index] + mywt) //update both cases where parent of myn == index or parent of myn != index
//					{
//						//if (SSSP[myn].EDGwt < mywt && SSSP[myn].Parent == index) //check if we have taken an edge with lower weight from the changeEdge set. if yes then don't update edgeweight
//						//{
//						//	mywt = SSSP[myn].EDGwt;
//						//}
//						d_UpdatedDist[myn] = d_UpdatedDist[index] + mywt;
//						SSSP[myn].Dist = d_UpdatedDist[index] + mywt;
//						SSSP[myn].Update = true;
//						SSSP[myn].Parent = index;
//						change_d[0] = 1;
//						/*printf("Not inf: index: %d edge weight: %f parent: %d dist: %f\n", index, SSSP[index].EDGwt, SSSP[index].Parent, SSSP[index].Dist);
//						printf("Not infmyn: %d myn weight: %f parent: %d dist: %f\n", myn, mywt, SSSP[myn].Parent, SSSP[myn].Dist);
//						*/
//						continue;
//					}
//					else
//					{
//						if (SSSP[myn].Parent == index)
//						{
//							d_UpdatedDist[myn] = d_UpdatedDist[index] + mywt;
//							SSSP[myn].Dist = d_UpdatedDist[index] + mywt;
//							SSSP[myn].Update = true;
//							/*SSSP[myn].Parent = index;*/ //Parent will remain same
//							change_d[0] = 1;
//							continue;
//						}
//						if ((d_UpdatedDist[index] > d_UpdatedDist[myn] + mywt) /*&& (SSSP[myn].Parent != index)*/)
//						{
//							d_UpdatedDist[index] = d_UpdatedDist[myn] + mywt;
//							SSSP[index].Dist = d_UpdatedDist[myn] + mywt;
//							SSSP[index].Update = true;
//							SSSP[index].Parent = myn;
//							change_d[0] = 1;
//						}
//					}
//				}
//			}
//		}
//	}
//}
//
//__global__ void checkIfDistUpdated(int X_size, double* d_OldUpdate, double* d_UpdatedDist, RT_Vertex* SSSP)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//	if (index < X_size)
//	{
//		if (d_OldUpdate[index] != d_UpdatedDist[index])
//		{
//			d_OldUpdate[index] = d_UpdatedDist[index];
//			SSSP[index].Update = true;
//		}
//		else { SSSP[index].Update = false; }
//	}
//}
//
//__global__ void updateDistanceFinal(int X_size, double* d_UpdatedDist, RT_Vertex* SSSP, double inf)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//	if (index < X_size)
//	{
//		int flag = 0;
//		//do not update parent
//		if (SSSP[index].Parent == -1) { flag = 1; }
//
//		if (flag == 0)
//		{
//			int px = SSSP[index].Parent;
//			if (px > -1)
//			{
//				//printf("XX %d :%d \n", i, px);  
//				if (index == SSSP[px].Parent)
//				{
//					printf("BBP %d %d \n", index, px);
//				}
//			}
//			if (d_UpdatedDist[index] >= inf)
//			{
//				SSSP[index].Dist = inf;
//			}
//			else
//			{
//				SSSP[index].Dist = d_UpdatedDist[SSSP[index].Parent] + SSSP[index].EDGwt;
//				//printf("Check 2. index: %d dist: %f, parent dist:%f, edgewt: %f \n", index, SSSP[index].Dist, d_UpdatedDist[SSSP[index].Parent], SSSP[index].EDGwt); //Test 23-01-2020
//			}
//		}
//	}
//}
//
//void edge_update(int* totalChange, int* X_size, int* SSSP_size, xEdge_cuda* allChange_cuda, Colwt2* cuda_adjlist_full_X, int* colStartPtr_X, RT_Vertex* SSSP, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R, int* te, int* nodes);
//void rest_update(int* X_size, Colwt2* cuda_adjlist_full_X, int* colStartPtr_X, RT_Vertex* SSSP, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R, int* nodes);
//
//
//int main(int argc, char* argv[]) {
//
//	double startx, endx, starty, endy;
//	/*double inf = std::numeric_limits<double>::infinity();*/
//
//	/***** Preprocessing to Graph (GUI) ***********/
//	int nodes;
//	printf("Enter number of total nodes: ");
//	scanf("%d", &nodes);
//	printf("check 0");
//
//
//
//	/*** Read Remainder Edges as Graph ***/
//	A_Network R;
//	/*R.reserve(nodes);*/
//	printf("check 1");
//	//The below 2 lines are creating segmentation fault
//	/*ADJ_Bundle aBundle;
//	R.resize(nodes, aBundle);*/
//	//uncomment below for loop in case of my files
//	/*for (int i = 0; i < nodes; i++)
//	{
//		ADJ_Bundle adj_bundle;
//		adj_bundle.Row = i;
//		R.push_back(adj_bundle);
//	}*/
//
//	//use below for direct path
//	/*string file1 = "C:\\Users\\khand\\Desktop\\PhD\\CUDA test\\Test\\test 1\\fullGraph.txt";
//	char* cstr1 = &file1[0];
//	readin_graphU(&R, nodes, cstr1);*/
//
//	//use below code if we use pass file name as argument
//	//readin_graphU(&R, nodes, argv[1]);
//
//
//	//use below code to pass the file name as relative path.
//	//**keep the files in the same folder
//	string file1 = "./fullGraph.txt";
//	char* cstr1 = &file1[0];
//	/*readin_graphU(&R, nodes, cstr1);*/
//	readin_graphU2(&R, nodes, cstr1);
//	/*readin_network2(&R, cstr1, -1);
//	*/
//	cout << "success" << endl;
//	/*for (int i = 0; i < R.size(); i++)
//	{
//		cout <<"*******"<< R.at(i).Row << endl;
//			for (auto y: R.at(i).ListW)
//			{
//				cout <<"node: "<< y.first <<"weight: "<< y.second << endl;
//			}
//	}*/
//
//	//checked1
//
//
//	cudaError_t cudaStatus;
//	/*int* key_R = new int[nodes];*/ //it stores the node. key is used to find the adj list of a specific node
//	int* colStartPtr_R;
//	cout << "success 1" << endl;
//	cudaStatus = cudaMallocManaged((void**) &colStartPtr_R, (nodes + 1) * sizeof(int)); //we take nodes +1 to store the start ptr of the first row 
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaMalloc failed!");
//		/*goto Error;*/
//	}
//	cout << "success 2" << endl;
//	int total_adjmatrix_size_R = 0;
//	cout << "success 2.5" << endl;
//	colStartPtr_R[0] = 0;
//	/*key_R[0] = R.at(0).Row;*/
//	for (int i = 0; i < R.size(); i++)
//	{
//		/*key_R[i] = R.at(i).Row;*/
//		int size = R.at(i).ListW.size();
//		colStartPtr_R[i + 1] = colStartPtr_R[i] + size; //size of adjacency list per row is stored
//		total_adjmatrix_size_R = total_adjmatrix_size_R + size;
//	}
//	cout << "success 3" << endl; //upto this is ok
//	Colwt2* cuda_adjlist_full_R;
//	cudaStatus = cudaMallocManaged(&cuda_adjlist_full_R, total_adjmatrix_size_R * sizeof(Colwt2));
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaMalloc failed!");
//		/*goto Error;*/
//	}
//	cout << "success 3.5" << endl;
//	for (int i = 0; i < R.size(); i++)
//	{
//		int colsize = R.at(i).ListW.size();
//
//		for (int j = 0; j < colsize; j++)
//		{
//			cuda_adjlist_full_R[colStartPtr_R[i] + j].col = R.at(i).ListW.at(j).first;
//			cuda_adjlist_full_R[colStartPtr_R[i] + j].wt = R.at(i).ListW.at(j).second;
//		}
//	}
//	////process 2: experimental
//	//int *cuda_adjlist_full_R_col, *d_cuda_adjlist_full_R_col;
//	//int *cuda_adjlist_full_R_wt, *d_cuda_adjlist_full_R_wt;
//	//cudaStatus = cudaMalloc((void**)&d_cuda_adjlist_full_R_col, total_adjmatrix_size_R * sizeof(int));
//	//if (cudaStatus != cudaSuccess) {
//	//	fprintf(stderr, "cudaMalloc failed 1");
//	//	/*goto Error;*/
//	//}
//	//cudaStatus = cudaMalloc((void**)&d_cuda_adjlist_full_R_wt, total_adjmatrix_size_R * sizeof(int));
//	//if (cudaStatus != cudaSuccess) {
//	//	fprintf(stderr, "cudaMalloc failed 2");
//	//	/*goto Error;*/
//	//}
//	//cuda_adjlist_full_R_col = (int*)malloc(total_adjmatrix_size_R * sizeof(int));
//	//cuda_adjlist_full_R_wt = (int*)malloc(total_adjmatrix_size_R * sizeof(int));
//	//for (int i = 0; i < R.size(); i++)
//	//{
//	//	int colsize = R.at(i).ListW.size();
//
//	//	for (int j = 0; j < colsize; j++)
//	//	{
//	//		cuda_adjlist_full_R_col[colStartPtr_R[i] + j] = R.at(i).ListW.at(j).first;
//	//		cuda_adjlist_full_R_wt[colStartPtr_R[i] + j] = (int)R.at(i).ListW.at(j).second;
//	//	}
//	//}
//	//cudaMemcpy(cuda_adjlist_full_R_col, d_cuda_adjlist_full_R_col, total_adjmatrix_size_R * sizeof(int), cudaMemcpyHostToDevice);
//	//cudaMemcpy(cuda_adjlist_full_R_wt, d_cuda_adjlist_full_R_wt, total_adjmatrix_size_R * sizeof(int), cudaMemcpyHostToDevice);
//	//cudaFree(d_cuda_adjlist_full_R_col); //test
//	//cudaFree(d_cuda_adjlist_full_R_wt); //test
//	//cout << "success 4" << endl;
//	////process 2 ends:
//	////checked2
//
//	///*** Finished Reading CRT Tree **/
//
//	/*** Read the Certificate ***/
//	
//	A_Network X;
//	/*ADJ_Bundle bBundle;
//	X.resize(nodes, bBundle);*/
//	for (int i = 0; i < nodes; i++)
//	{
//		ADJ_Bundle adj_bundle;
//		adj_bundle.Row = i;
//		X.push_back(adj_bundle);
//	}
//
//	//use below for direct path
//	/*string file2 = "C:\\Users\\khand\\Desktop\\PhD\\CUDA test\\Test\\test 1\\SSSP.txt";
//	char* cstr2 = &file2[0];
//	readin_network(&X, cstr2, -1);*/
//
//	//use below code if we use pass file name as argument
//	/*readin_network(&X, argv[2], -1);*/
//
//	//use below code to pass the file name as relative path.
//	//**keep the files in the same folder
//	string file2 = "./SSSP.txt";
//	char* cstr2 = &file2[0];
//	readin_network(&X, cstr2, -1);
//
//
//	int* key_X = new int[nodes]; //it stores the node. key is used to find the adj list of a specific node
//	int* colStartPtr_X = new int[nodes + 1]; //we take nodes +1 to store the start ptr of the first row 
//	int total_adjmatrix_size_X = 0;
//	colStartPtr_X[0] = 0;
//	key_X[0] = X.at(0).Row;
//	for (int i = 0; i < X.size(); i++)
//	{
//		key_X[i] = X.at(i).Row;
//		int size = X.at(i).ListW.size();
//		colStartPtr_X[i + 1] = colStartPtr_X[i] + size; //size of adjacency list per row is stored
//		total_adjmatrix_size_X = total_adjmatrix_size_X + size;
//	}
//
//	Colwt2* cuda_adjlist_full_X;
//	cudaMallocManaged(&cuda_adjlist_full_X, total_adjmatrix_size_X * sizeof(Colwt2));
//	for (int i = 0; i < X.size(); i++)
//	{
//		int colsize = X.at(i).ListW.size();
//
//		for (int j = 0; j < colsize; j++)
//		{
//			cuda_adjlist_full_X[colStartPtr_X[i] + j].col = X.at(i).ListW.at(j).first;
//			cuda_adjlist_full_X[colStartPtr_X[i] + j].wt = X.at(i).ListW.at(j).second;
//			/*cout <<"col: "<< cuda_adjlist_full_X[colStartPtr_X[i] + j].col << " weight: " << cuda_adjlist_full_X[colStartPtr_X[i] + j].wt << endl;*/
//		}
//	}
//
//	/*** Finished Reading CRT Tree **/
//
//	//List of Changes
//	//There will be a list for inserts and a list for delete
//	vector<xEdge> allChange;
//	allChange.clear();
//
//	/*** Read set of Changed Edges ***/
//   //use below for direct path
//   /*string file3 = "C:\\Users\\khand\\Desktop\\PhD\\CUDA test\\Test\\test 1\\changeEdges.txt";
//   char* cstr3 = &file3[0];
//   readin_changes(cstr3, &allChange);*/
//
//   //use below code if we use pass file name as argument
//   /*readin_changes(argv[3], &allChange);*/
//
//   //use below code to pass the file name as relative path.
//	string file3 = "./changeEdges.txt";
//	char* cstr3 = &file3[0];
//	readin_changes(cstr3, &allChange);
//
//	//new addition
//	xEdge_cuda* allChange_cuda;
//	int totalChange = allChange.size();
//	cudaMallocManaged(&allChange_cuda, totalChange * sizeof(xEdge_cuda));
//	for (int i = 0; i < totalChange; i++)
//	{
//		allChange_cuda[i].node1 = allChange.at(i).theEdge.node1;
//		allChange_cuda[i].node2 = allChange.at(i).theEdge.node2;
//		allChange_cuda[i].edge_wt = allChange.at(i).theEdge.edge_wt;
//		allChange_cuda[i].inst = allChange.at(i).inst;
//		allChange_cuda[i].insertedToDatastructure = allChange.at(i).insertedToDatastructure;
//	}
//	/*** Finished Reading Changed Edges **/
//
//	//Initializing  Rooted Tree
//	RT_Vertex* SSSP;
//	cudaMallocManaged(&SSSP, nodes * sizeof(RT_Vertex));
//	int* stencil; //stencil is used for tracking which node is being affected. 
//	/*cudaMallocManaged(&stencil, nodes * sizeof(int));*/
//	cudaMalloc((void**)&stencil, nodes * sizeof(int));
//	int* stencil_c = new int[nodes];
//	/*vector<SCC_vertex>SCC;*/
//	int graphDirectedUndirectedIndicator = 0; // Should be 1 for SCC, 0 for not SCC. need to modify if we want SCC
//
//	int source;
//	printf("Enter source node: ");
//	scanf("%d", &source);
//	int p;
//
//
//
//	if (graphDirectedUndirectedIndicator == 0) {
//		int src = source; //the source from which the paths are computed
//		initialize << <(nodes / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (nodes, src, SSSP, stencil); //kernet call
//		cudaDeviceSynchronize();
//		cudaMemcpy(stencil_c, stencil, nodes * sizeof(int), cudaMemcpyDeviceToHost);
//		/*for (int i = 0; i < nodes; i++)
//		{
//			cout << "stencil_c" << stencil_c[i] << endl;
//		}*/
//		/*for (int i = 0; i < nodes; i++)
//		{
//
//			cout <<"dist"<< SSSP->Dist << endl;
//			cout <<"wt"<< SSSP->EDGwt << endl;
//			cout << "level"<< SSSP->Root << endl;
//			cout << "marked"<< SSSP->Parent << endl;
//		}*/
//
//
//		//Code for create_tree:
//		//Time calculation
//		auto startTime = high_resolution_clock::now();
//		int totalAffectedNode; //alias of numberOfAffectedNode
//
//		int* affectedPointer;
//		int* d_affectedPointer;
//		cudaMalloc((void**)&d_affectedPointer, nodes * sizeof(int));
//		affectedPointer = (int*)calloc(nodes, sizeof(int));
//		cudaMemcpy(d_affectedPointer, affectedPointer, nodes * sizeof(int), cudaMemcpyHostToDevice);
//		/*cudaMallocManaged(&affectedPointer, nodes * sizeof(int));*/
//
//		totalAffectedNode = 1;
//		int start = 0, end = 0;
//		int* affected_nodes;
//		affected_nodes = (int*)calloc(totalAffectedNode, sizeof(int));
//		affected_nodes[0] = src;
//		cudaStream_t stream1;
//		cudaError_t result;
//		result = cudaStreamCreate(&stream1);
//		while (totalAffectedNode > 0)
//		{
//			vector<int> affectedNodeAlias;
//			/*ThreadHelper* threadHelpers;
//			cudaMallocManaged(&threadHelpers, (totalAffectedNode+1) * sizeof(ThreadHelper));
//			int offset = 0;
//			int totalNumberofCudaThread = 0;*/
//
//			for (int i = 0; i < totalAffectedNode; i++)
//			{
//				affectedNodeAlias.push_back(affected_nodes[i]);
//			}
//			for (int i = 0; i < totalAffectedNode; i++)
//			{
//				p = affectedNodeAlias.at(i);
//				src = p;
//				/*cout << "src: " << src << endl;*/
//				start = colStartPtr_X[p];
//				end = colStartPtr_X[p + 1];
//				int numberofCudaThread = end - start;
//				/*threadHelpers[i].offset = offset;
//				offset = offset + numberofCudaThread;*/
//
//				/*cout << "adj node 4:" << cuda_adjlist_full_X[4].col<<endl;
//				for (int i = 0; i < numberofCudaThread; i++)
//				{
//					cout << "adj node ptr" << i + start << endl;
//					cout << "adj node"<<cuda_adjlist_full_X[i + start].col<<endl;
//					cout << "adj node"<<cuda_adjlist_full_X[i + start].col<<endl;
//				}*/
//				//create_tree method creates the SSSP tree with values stored in cuda_adjlist_full_X.
//				//This SSSP tree is the input SSSP tree
//				create_tree << <(numberofCudaThread / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK, 0, stream1 >> > (cuda_adjlist_full_X, start, SSSP, src, d_affectedPointer, numberofCudaThread);
//
//				/*threadHelpers[i].src = p;
//				threadHelpers[i].start = start;*/
//			}
//			/*threadHelpers[totalAffectedNode].offset = offset;
//			totalNumberofCudaThread = offset;
//			create_tree2<<<(offset / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>> > (cuda_adjlist_full_X, SSSP, d_affectedPointer, threadHelpers, totalNumberofCudaThread);
//			cudaDeviceSynchronize();*/
//			cudaStreamSynchronize(stream1);
//			thrust::device_ptr<int> affectedPointer_alias(d_affectedPointer); // converting raw ptr to device_ptr
//			cudaMemcpy(affectedPointer, d_affectedPointer, nodes * sizeof(int), cudaMemcpyDeviceToHost);
//			//Test code - start
//			/*for (int i = 0; i < nodes; i++)
//			{
//				cout << "after kernel call:" << affectedPointer[i] << endl;
//			}*/
//			//Test code - end
//			/*thrust::device_ptr<int> affectedPointer_alias(affectedPointer);*/
//			thrust::device_vector<int> affectedPointer_vector(affectedPointer_alias, affectedPointer_alias + nodes); //converting device_ptr to device_vector
//			totalAffectedNode = thrust::count(affectedPointer_vector.begin(), affectedPointer_vector.end(), 1); //count the number of affected node
//			//test code start
//			/*cout << "totalAffectedNode: " << totalAffectedNode<<endl;*/
//			//test code end
//			affected_nodes = (int*)realloc(affected_nodes, totalAffectedNode * sizeof(int));
//			/*affectedPointer = thrust::raw_pointer_cast(&affectedPointer_vector[0]);*/
//			thrust::copy_if(thrust::host, stencil_c, stencil_c + nodes, affectedPointer, affected_nodes, is_affected());
//			//Test code - start
//			/*cout << "affected nodes " << endl;
//			for (int i = 0; i < totalAffectedNode; i++)
//			{
//				cout << affected_nodes[i] << endl;
//			}*/
//			//Test code - end
//			affectedPointer = (int*)calloc(nodes, sizeof(int));
//			cudaMemcpy(d_affectedPointer, affectedPointer, nodes * sizeof(int), cudaMemcpyHostToDevice);
//			/*cudaFree(threadHelpers);*/
//		}
//		free(affectedPointer);
//		free(affected_nodes);
//		cudaFree(d_affectedPointer);
//
//		//Time calculation
//		auto stopTime = high_resolution_clock::now();
//		// Time calculation
//		auto duration = duration_cast<microseconds>(stopTime - startTime);
//		cout << "Time taken by create-tree function: "
//			<< duration.count() << " microseconds" << endl;
//
//		//test
//		/*cout << "input sssp tree" << endl;
//		for (int i = 0; i < nodes; i++)
//		{
//			cout << "node" << i << endl;
//			cout << "dist" << SSSP[i].Dist << endl;
//			cout << "parent" << SSSP[i].Parent << endl;
//		}*/
//		//test end
//		//edge_update function
//		//Update the inserted and delted edges in the tree
//		int x_size = X.size();
//		int SSSP_size = nodes; //considering all nodes are participating in the SSSP
//		int te = 0;
//		auto startTime1 = high_resolution_clock::now();
//		edge_update(&totalChange, &x_size, &SSSP_size, allChange_cuda, cuda_adjlist_full_X, colStartPtr_X, SSSP, cuda_adjlist_full_R, colStartPtr_R, &te, &nodes);
//		cout << "after edge_update fn" << endl;
//		//Time calculation
//		auto stopTime1 = high_resolution_clock::now();
//		// Time calculation
//		auto duration1 = duration_cast<microseconds>(stopTime1 - startTime1);
//		cout << "Time taken by edge_update function: "
//			<< duration1.count() << " microseconds" << endl;
//
//		auto startTime2 = high_resolution_clock::now();
//		rest_update(&x_size, cuda_adjlist_full_X, colStartPtr_X, SSSP, cuda_adjlist_full_R, colStartPtr_R, &nodes);
//		cout << "after rest_update fn" << endl;
//		//Time calculation
//		auto stopTime2 = high_resolution_clock::now();
//		// Time calculation
//		auto duration2 = duration_cast<microseconds>(stopTime2 - startTime2);
//		cout << "Time taken by rest_update function: "
//			<< duration1.count() << " microseconds" << endl;
//	}
//	else
//	{
//		//****below code needs modification
//		/*SCC.clear();
//		readin_SCC(argv[2], &SCC);
//		update_SCC(&X, &SCC, &allChange);
//		print_network(X);*/
//	}
//	//Test code start
//	cout << "SSSP" << endl;
//	for (int i = 0; i < nodes; i++)
//	{
//		cout << "*******" << endl;
//		cout << "node" << i << endl << "dist" << SSSP[i].Dist << endl << "parent" << SSSP[i].Parent << endl;
//	}
//	//Test code end
//
//
//	cudaFree(colStartPtr_R);
//	cudaFree(cuda_adjlist_full_R);
//	cudaFree(colStartPtr_X);
//	cudaFree(cuda_adjlist_full_X);
//	cudaFree(allChange_cuda);
//	cudaFree(SSSP);
//	cudaFree(stencil);
//	Error:
//		cudaFree(colStartPtr_R);
//	return 0;
//}
//
//void edge_update(int* totalChange, int* X_size, int* SSSP_size, xEdge_cuda* allChange_cuda, Colwt2* cuda_adjlist_full_X, int* colStartPtr_X, RT_Vertex* SSSP, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R, int* te, int* nodes)
//{
//	double inf = std::numeric_limits<double>::infinity();
//	/*int* Edgedone;*/
//	double* UpdatedDist;
//
//	int iter = 0;
//
//	//Mark how the edge is processed
//	int* Edgedone;
//	cudaMalloc((void**)&Edgedone, (*totalChange) * sizeof(int));
//	//initialize Edgedone array with -1
//	initializeEdgedone << <((*totalChange) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (Edgedone, *totalChange);
//	cudaDeviceSynchronize();
//
//	/*thrust::device_vector<int> Edgedone_ptr(*totalChange);
//	thrust::fill(Edgedone_ptr.begin(), Edgedone_ptr.end(), -1);
//	int* Edgedone = thrust::raw_pointer_cast(Edgedone_ptr);*/
//
//	//Store the updated distance value
//	UpdatedDist = (double*)calloc(*X_size, sizeof(double));
//	double* d_UpdatedDist;
//	cudaMalloc((void**)&d_UpdatedDist, (*X_size) * sizeof(double));
//	cudaMemcpy(d_UpdatedDist, UpdatedDist, (*X_size) * sizeof(double), cudaMemcpyHostToDevice);
//
//	//Initialize with current distance for each node
//	initializeUpdatedDist << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (d_UpdatedDist, SSSP, *X_size);
//	cudaDeviceSynchronize();
//	/*	cudaMemcpy(UpdatedDist, d_UpdatedDist, (*X_size) * sizeof(double), cudaMemcpyDeviceToHost);*/ //not required
//
//
//	int numS = *totalChange;
//	int* d_colStartPtr_X;
//	cudaMalloc((void**)&d_colStartPtr_X, (*nodes + 1) * sizeof(int));
//	cudaMemcpy(d_colStartPtr_X, colStartPtr_X, (*nodes + 1) * sizeof(int), cudaMemcpyHostToDevice);
//
//	insertDeleteEdge << < (numS / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (allChange_cuda, Edgedone, SSSP, numS, *X_size, d_colStartPtr_X, cuda_adjlist_full_X, d_UpdatedDist, inf, cuda_adjlist_full_R, colStartPtr_R);
//	cudaDeviceSynchronize();
//
//
//	/*int* Edgedone_c = new int[*totalChange];
//	cudaMemcpy(Edgedone_c, Edgedone, *totalChange * sizeof(int), cudaMemcpyDeviceToHost); *///not req.
//	/*cudaMemcpy(UpdatedDist, d_UpdatedDist, (*X_size) * sizeof(double), cudaMemcpyDeviceToHost); *///not req.
//
//
//	//Go over the inserted edges to see if they need to be changed
//	int* change_d = new int[1];
//	int* change = new int[1];
//	change[0] = 1;
//	cudaMalloc((void**)&change_d, 1 * sizeof(int));
//	cudaMemcpy(change_d, change, 1 * sizeof(int), cudaMemcpyHostToDevice);
//	while (change[0] == 1)
//	{
//		change[0] = 0;
//		cudaMemcpy(change_d, change, 1 * sizeof(int), cudaMemcpyHostToDevice);
//		checkInsertedEdges << < (numS / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (numS, Edgedone, d_UpdatedDist, allChange_cuda, SSSP, change_d);
//		cudaDeviceSynchronize();
//		cudaMemcpy(change, change_d, 1 * sizeof(int), cudaMemcpyDeviceToHost);
//		/*cout << "change"<< change[0]<<endl;*/
//
//	}
//
//	//Update the distances
//	 //Initialize with current distance for each node
//	updateDistance << < (numS / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (*X_size, SSSP, d_UpdatedDist, inf);
//	cudaDeviceSynchronize();
//
//
//	cudaFree(change_d);
//	cudaFree(d_UpdatedDist);
//	cudaFree(d_colStartPtr_X);
//	free(UpdatedDist);
//	return;
//}
//
//
//void rest_update(int* X_size, Colwt2* cuda_adjlist_full_X, int* colStartPtr_X, RT_Vertex* SSSP, Colwt2* cuda_adjlist_full_R, int* colStartPtr_R, int* nodes)
//{
//	double inf = std::numeric_limits<double>::infinity();
//
//
//	int its = 0; //number of iterations
//
//	int* change_d = new int[1];
//	int* change = new int[1]; //marking whether the connections changed in the iteration
//	change[0] = 1;
//	cudaMalloc((void**)&change_d, 1 * sizeof(int));
//	cudaMemcpy(change_d, change, 1 * sizeof(int), cudaMemcpyHostToDevice);
//
//	double* UpdatedDist;
//	//Store the updated distance value
//	UpdatedDist = (double*)calloc(*X_size, sizeof(double));
//	double* d_UpdatedDist;
//	cudaMalloc((void**)&d_UpdatedDist, (*X_size) * sizeof(double));
//	cudaMemcpy(d_UpdatedDist, UpdatedDist, (*X_size) * sizeof(double), cudaMemcpyHostToDevice);
//
//
//	double* OldUpdate;
//	//Store the old updated distance value
//	OldUpdate = (double*)calloc(*X_size, sizeof(double));
//	double* d_OldUpdate;
//	cudaMalloc((void**)&d_OldUpdate, (*X_size) * sizeof(double));
//	cudaMemcpy(d_OldUpdate, OldUpdate, (*X_size) * sizeof(double), cudaMemcpyHostToDevice);
//
//
//	int* mychange;
//	//Store the old updated distance value
//	mychange = (int*)calloc(*X_size, sizeof(int));
//	int* d_mychange;
//	cudaMalloc((void**)&d_mychange, (*X_size) * sizeof(int));
//	cudaMemcpy(d_mychange, mychange, (*X_size) * sizeof(int), cudaMemcpyHostToDevice);
//
//	int* d_colStartPtr_X;
//	cudaMalloc((void**)&d_colStartPtr_X, (*nodes + 1) * sizeof(int));
//	cudaMemcpy(d_colStartPtr_X, colStartPtr_X, (*nodes + 1) * sizeof(int), cudaMemcpyHostToDevice);
//
//
//	//Initialize with current distance for each node
//	initializeUpdatedDistOldDist << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (d_UpdatedDist, d_OldUpdate, SSSP, *X_size);
//	cudaDeviceSynchronize();
//
//
//	int iter = 0;
//	while (change[0] == 1 && its < 70)
//	{
//		printf("Iteration:%d \n", its);
//
//		change[0] = 0;
//		cudaMemcpy(change_d, change, 1 * sizeof(int), cudaMemcpyHostToDevice);
//		updateNeighbors << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (d_UpdatedDist, SSSP, *X_size, d_mychange, d_colStartPtr_X, cuda_adjlist_full_X, inf, change_d, its, cuda_adjlist_full_R, colStartPtr_R);
//		cudaDeviceSynchronize();
//		cudaMemcpy(change, change_d, 1 * sizeof(int), cudaMemcpyDeviceToHost);
//
//		//Test code start
//		/*cudaMemcpy(UpdatedDist, d_UpdatedDist, (*X_size) * sizeof(double), cudaMemcpyDeviceToHost);
//		for (int i = 0; i < *X_size; i++)
//		{
//			cout << "UpdatedDist: " << UpdatedDist[i] << endl;
//		}*/
//		//Test code end
//
//
//	//Check if distance was updated
//		checkIfDistUpdated << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (*X_size, d_OldUpdate, d_UpdatedDist, SSSP);
//		cudaDeviceSynchronize();
//		its++;
//	}//end of while
//	printf("Total Iterations to Converge %d \n", its);
//
//	//Update the distances
//	//Initialize with current distance for each node
//	updateDistanceFinal << <((*X_size) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (*X_size, d_UpdatedDist, SSSP, inf);
//	cudaDeviceSynchronize();
//
//
//	free(UpdatedDist);
//	free(OldUpdate);
//	free(mychange);
//	cudaFree(change_d);
//	cudaFree(d_UpdatedDist);
//	cudaFree(d_OldUpdate);
//	cudaFree(d_mychange);
//	cudaFree(d_colStartPtr_X);
//
//	return;
//}
//
